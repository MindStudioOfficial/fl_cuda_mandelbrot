#include "hip/hip_runtime.h"
#include <render.cuh>

int w, h;
double r1, r2, i1, i2;
bool setup = false;
uint64_t *d_iters = nullptr;
uint8_t *img = nullptr, *d_img = nullptr;

uint64_t max_iter = 0;

__device__ Complex_t operator+(Complex_t a, Complex_t b)
{
    Complex_t c = {a.re + b.re, a.im + b.im};
    return c;
}

__device__ Complex_t operator*(Complex_t a, Complex_t b)
{
    Complex_t c = {a.re * b.re - a.im * b.im, a.re * b.im + a.im * b.re};
    return c;
}

__device__ double complexNoRootAbs(Complex_t a)
{
    return a.re * a.re + a.im * a.im;
}

__device__ double cabs(Complex_t c)
{
    return sqrt(c.re * c.re + c.im * c.im);
}

__device__ uchar3 hsvToRGB(float3 hsv)
{
    float f = hsv.x / 60.0f;
    float hi = floorf(f);
    f = f - hi;
    float p = hsv.z * (1 - hsv.y);
    float q = hsv.z * (1 - hsv.y * f);
    float t = hsv.z * (1 - hsv.y * (1 - f));

    float r, g, b;

    if (hi == 0.0f || hi == 6.0f)
    {
        r = hsv.z;
        g = t;
        b = p;
    }
    else if (hi == 1.0f)
    {
        r = q;
        g = hsv.z;
        b = p;
    }
    else if (hi == 2.0f)
    {
        r = p;
        g = hsv.z;
        b = t;
    }
    else if (hi == 3.0f)
    {
        r = p;
        g = q;
        b = hsv.z;
    }
    else if (hi == 4.0f)
    {
        r = t;
        g = p;
        b = hsv.z;
    }
    else
    {
        r = hsv.z;
        g = p;
        b = q;
    }

    unsigned char red = __float2uint_rn(255.0f * r);
    unsigned char green = __float2uint_rn(255.0f * g);
    unsigned char blue = __float2uint_rn(255.0f * b);

    return make_uchar3(red, green, blue);
}

__global__ void kernelIterate(int w, int h, double r1, double r2, double i1, double i2, uint64_t *d_iters, int max_iters)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= w || idy >= h)
        return;

    int pix = idy * w + idx;

    double real = r1 + ((r2 - r1) / w) * idx;
    double imag = i1 + ((i2 - i1) / h) * idy;

    Complex_t z = {0, 0};
    Complex_t c = {real, imag};

    int i = 0;
    while (complexNoRootAbs(z) <= 4.0 && i < max_iters)
    {
        z = z * z + c;
        i++;
    }
    d_iters[pix] = i;
}

__global__ void kernelDraw(int width, int height, uint64_t *d_iters, uint8_t *d_img, uint64_t maxiter)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= width || idy >= height)
        return;

    int pix = idy * width + idx;

    int64_t i = d_iters[pix];

    float h = (i % 360);
    float s = 1.f;
    float v = 1.f;

    if (i >= maxiter - 1)
        v = 0.f;

    uchar3 rgb = hsvToRGB(make_float3(h, s, v));

    d_img[pix * 4 + 3] = 255;
    d_img[pix * 4 + 0] = rgb.x;
    d_img[pix * 4 + 1] = rgb.y;
    d_img[pix * 4 + 2] = rgb.z;
}

EXTERNC void iterate(int max_iters)
{
    if (!setup)
    {
        printf("call setupCanvas before iterating!\n");
        return;
    }
    max_iter = max_iters;

    int pixs = w * h;

    int bw = THREADS;
    dim3 bs = dim3(16, 16);
    dim3 gs = dim3((w + bs.x - 1) / bs.x, (h + bs.y - 1) / bs.y);

    kernelIterate<<<gs, bs>>>(w, h, r1, r2, i1, i2, d_iters, max_iters);

    hipDeviceSynchronize();
}

EXTERNC void draw()
{
    if (!setup)
    {
        printf("call setupCanvas before drawing!\n");
        return;
    }

    int pixs = w * h;
    dim3 bs = dim3(16, 16);
    dim3 gs = dim3((w + bs.x - 1) / bs.x, (h + bs.y - 1) / bs.y);

    kernelDraw<<<gs, bs>>>(w, h, d_iters, d_img, max_iter);

    hipDeviceSynchronize();

    size_t imgsize = sizeof(uint8_t) * pixs * 4;

    hipMemcpy(img, d_img, imgsize, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
}

EXTERNC void dispose()
{
    if (d_iters != nullptr)
    {
        hipFree(d_iters);
        d_iters = nullptr;
    }
    if (d_img != nullptr)
    {
        hipFree(d_img);
        d_img = nullptr;
    }
    setup = false;
}

EXTERNC uint8_t *setupCanvas(int fWidth, int fHeight, double re1, double re2, double im1, double im2)
{
    dispose();
    w = fWidth;
    h = fHeight;
    r1 = re1;
    r2 = re2;
    i1 = im1;
    i2 = im2;

    size_t itersize = sizeof(uint64_t) * w * h;

    hipMalloc(&d_iters, itersize);
    hipMemset(d_iters, 0, itersize);

    size_t imgsize = sizeof(uint8_t) * w * h * 4;

    hipMalloc(&d_img, imgsize);
    hipMemset(d_img, 0, imgsize);

    img = (uint8_t *)calloc(1, imgsize);

    size_t valssize = sizeof(Complex_t) * w * h;

    hipDeviceSynchronize();

    setup = true;

    return img;
}
